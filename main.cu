
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <sstream>
#include <fstream>
#include <numeric>
#include <stdlib.h>
#include <vector>
#include <algorithm>
using namespace std;
#define REDUCE_BLOCK_SIZE 128

struct Matrix {
	Matrix() : elements(NULL), width(0), height(0), pitch(0) {}
	~Matrix() { if (elements) delete[] elements; }
	unsigned int width;
    unsigned int height;
    unsigned int pitch;
    float* elements;
};

__global__ void kernelMatrixMul(float*, float*, float*, int, int, int, int);
__global__ void kernelSigmoid(float*, int);
__global__ void kernelMatrixAbsError(float*, float*, float*, int, int);
__global__ void kernelAbsError(float*, float*, float*, int);
__global__ void kernelUpdateParams(float*, float*, float*, float*, int, float);
__global__ void kernelCrossEntropy(float*, float*, float*, int);
__global__ void kernelReduction(float*, float*, int);
__global__ void confusion_matrix_kernel(float*, float*, int*, int, int);

inline static void InitializeMatrix(Matrix *mat, int x, int y, float val) {
	if (x > mat->width || y > mat->height) {
		throw ("invalid access - Initialize Matrix");
	}
	mat->elements[y * mat->width + x] = val;
}

inline static float Matrix_Element_Required(Matrix *mat, int x, int y)
{
	if (x > mat->width || y > mat->height) {
		throw ("invalid access - Matrix Element Required");
	}
	return mat->elements[y * mat->width + x];
}

static void AllocateMatrix(Matrix *mat, int height, int width)
{
	mat->elements = new float[height * width];
	mat->width = width;
	mat->height = height;
	for (int i = 0; i < mat->width; i++) {
		for (int j = 0; j < mat->height; j++) {
			InitializeMatrix(mat, i, j, 0.0f);
		}
	}
}

static void DisplayMatrix(Matrix &mat, bool force = false)
{
	std::cout << "Dim: " << mat.height << ", " << mat.width << "\n";
	if ((mat.width < 10 && mat.height < 10) || force)
	{
		for (int j = 0; j < mat.height; j++) {
			for (int i = 0; i < mat.width; i++) {
				std::cout << Matrix_Element_Required(&mat, i, j) << "\t";
			}
			std::cout << "\n";
		}
	}
	std::cout << std::endl;
}

static bool readData(string file_name, Matrix *X, Matrix *y) {

	ifstream s(file_name.c_str());
	//ifstream s(file_name);
	if (!s.is_open()) {
		//throw runtime_error(file_name + " doesn't exist");
		printf("The file does not exist\n");
	}

	int rows = 0;
	int cols = 0;
	string line;
	while (getline(s, line)) {
		// if we read first line, check how many columns
		if (rows++ == 0) {
			stringstream ss(line);

			while (ss.good()) {
				string substr;
				getline(ss, substr, ',');
				cols++;
			}
		}
	}
	std::cout << "Found " << rows << " rows with " << cols << " columns." << std::endl;
	s.clear() ;
	s.seekg(0, ios::beg);

	AllocateMatrix (X, rows - 1,cols - 2);
	AllocateMatrix (y, rows - 1, 1);

	// go to second line
	getline(s, line);
	int ya = 0;
	while (getline(s, line)) {
		stringstream ss(line);

		int xa = 0;
		while (ss.good()) {
			string substr;
			getline(ss, substr, ',');
			// first column is uninteresting
			// second column is target values
			if (xa == 1) {
				float val = atof(substr.c_str());
				InitializeMatrix(y, 0, ya, val);
			} else if (xa > 1) {
				float val = atof(substr.c_str());
				InitializeMatrix(X, (xa - 2), ya, val);
			}
			xa++;
		}
		ya++;
	}

	return true;
}

static void minMaxNormalization(Matrix *m)
{
	for (int x = 0; x < m->width; ++x) {
		// calculate std for each column
		float min = Matrix_Element_Required(m, x, 0);
		float max = Matrix_Element_Required(m, x, 0);
		for (int y = 1; y < m->height; ++y) {
			float val = Matrix_Element_Required(m, x, y);
			if (val < min) {
				min = val;
			} else if (val > max) {
				max = val;
			}
		}

		for (int y = 0; y < m->height; ++y) {
			float val = Matrix_Element_Required(m, x, y);
			InitializeMatrix(m, x, y, (val - min) / max);
		}
	}
}

static void InitializeRandom(Matrix *mat, float LO, float HI)
{
	for (int i = 0; i < mat->width; ++i) {
		for (int j = 0; j < mat->height; ++j) {
			float r = LO + static_cast <float> (rand()) /( static_cast <float> (RAND_MAX/(HI-LO)));
			InitializeMatrix(mat, i, j, r);
		}
	}
}

static void CheckCudaErrorAux (const char *file, unsigned line, const char *statement, hipError_t err)
{
	if (err == hipSuccess)
		return;
	std::cerr << statement<<" returned " << hipGetErrorString(err) << "("<<err<< ") at "<<file<<":"<<line << std::endl;
	exit (1);
}
#define SAFE_CALL(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

__global__ void kernelMatrixMul(float *m1, float *m2, float *r, int m1w, int m2w, int rw, int rh)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if ((row < rh) && (col < rw)) {
		// dot product
		float accum = 0.0f;
		for (int c = 0; c < m1w; c++)
		{
			float v1 = m1[row * m1w + c];
			float v2 = m2[c * m2w + col];
			accum += (v1 *  v2);
		}

		r[row * rw + col] = accum;
	}
}

__global__ void kernelSigmoid(float *r, int m)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < m) {
		float val = r[index];
		r[index] = 1.0 / (1.0 + expf(-val));
	}
}

__global__ void kernelMatrixAbsError(float *p, float *ys, float *r, int rw, int rh)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if ((row < rh) && (col < rw)) {
		float pval = p[row * rw + col];
		float ysval = ys[row * rw + col];

		float v = pval - ysval;
		r[row * rw + col] = v * v;
	}
}

__global__ void kernelAbsError(float *p, float *ys, float *r, int m)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < m) {
		float pval = p[index];
		float ysval = ys[index];

		float v = pval - ysval;
		r[index] = v * v;
	}
}

__global__ void kernelUpdateParams(float *p, float *ys, float *th, float *xs, int m, float alpha)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < m) {
		float h = *p;
		float y = *ys;

		float x = xs[index];

		th[index] = th[index] - alpha * (h - y) * x;
	}
}

__global__ void kernelCrossEntropy(float *p, float *ys, float *r, int m)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < m) {
		float pval = p[index];
		float ysval = ys[index];

		float ex = log1pf(expf(-ysval * pval));
		r[index] = ex;
	}
}

__global__ void kernelReduction(float * input, float * output, int len) {
    //@@ Load a segment of the input vector into shared memory
    __shared__ float partialSum[2 * REDUCE_BLOCK_SIZE];
    unsigned int t = threadIdx.x, start = 2 * blockIdx.x * REDUCE_BLOCK_SIZE;
    if (start + t < len)
       partialSum[t] = input[start + t];
    else
       partialSum[t] = 0;
    if (start + REDUCE_BLOCK_SIZE + t < len)
       partialSum[REDUCE_BLOCK_SIZE + t] = input[start + REDUCE_BLOCK_SIZE + t];
    else
       partialSum[REDUCE_BLOCK_SIZE + t] = 0;
    //@@ Traverse the reduction tree
    for (unsigned int stride = REDUCE_BLOCK_SIZE; stride >= 1; stride >>= 1) {
       __syncthreads();
       if (t < stride)
          partialSum[t] += partialSum[t+stride];
    }
    // Write the computed sum of the block to the output vector at the correct index
    if (t == 0)
       output[blockIdx.x] = partialSum[0];
}

__global__ void confusion_matrix_kernel(float* d_predictions, float* d_labels, int* d_confusion_matrix, int num_classes, int num_samples) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
	//int label;
    if (idx < num_samples) {
		int pred = (d_predictions[idx] >= 0.5) ? 1 : 0;;
        int label = static_cast<int>(d_labels[idx]);
       if (pred < num_classes && label < num_classes) {
            atomicAdd(&d_confusion_matrix[label * num_classes + pred], 1);
        }
    }
}

void calculate_confusion_matrix(float* d_predictions, float* d_labels, int* d_confusion_matrix, int num_classes, int num_samples) {
    // Initialize confusion matrix to zero on the GPU
    hipMemset(d_confusion_matrix, 0, num_classes * num_classes * sizeof(int));
	
    // Launch kernel to compute confusion matrix
    int threadsPerBlock = 256;
    int blocksPerGrid = (num_samples + threadsPerBlock - 1) / threadsPerBlock;

    confusion_matrix_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_predictions, d_labels, d_confusion_matrix, num_classes, num_samples);

    // Synchronize to ensure all threads are done
    hipDeviceSynchronize();
}


static void LogisticRegression(Matrix *X, Matrix *y, Matrix *Parameters, Matrix *Train_Parameters, int maxIterations, float alpha, vector<float> &cost_function, int num_classes)
{
	// put data into gpu
	float *gpu_X;
	float *gpu_y;

	float *gpu_prediction;

	float *gpu_params;
	float *gpu_abs_error;
	float *gpu_err_cost;

	float *gpu_predictions;
	int* d_confusion_matrix;
	Matrix predictions;
	AllocateMatrix(&predictions, y->height, y->width);

	Matrix absErrors;
	AllocateMatrix(&absErrors, y->height, y->width);

	float mean_error;
	float sum=0;
	int quantity = 1;

	int m = y->height;

	int numOutputElements;
	numOutputElements = m / (REDUCE_BLOCK_SIZE<<1);
	if (m % (REDUCE_BLOCK_SIZE<<1)) {
		numOutputElements++;
	}

	SAFE_CALL(hipMalloc((void**)&gpu_X, sizeof(float) * X->width * X->height));
	SAFE_CALL(hipMalloc((void**)&gpu_y, sizeof(float) * y->width * y->height));
	SAFE_CALL(hipMalloc((void**)&gpu_prediction, sizeof(float)));
	SAFE_CALL(hipMalloc((void**)&gpu_predictions, sizeof(float) * y->width * y->height));
	SAFE_CALL(hipMalloc((void**)&gpu_abs_error, sizeof(float) * y->width * y->height));
	SAFE_CALL(hipMalloc((void**)&gpu_params, sizeof(float) * Parameters->width * Parameters->height));
	SAFE_CALL(hipMalloc((void**)&gpu_err_cost, sizeof(float) * numOutputElements));
	SAFE_CALL(hipMalloc(&d_confusion_matrix, num_classes * num_classes * sizeof(int)));

	SAFE_CALL(hipMemcpy(gpu_X, X->elements, sizeof(float) * X->width * X->height, hipMemcpyHostToDevice));
	SAFE_CALL(hipMemcpy(gpu_y, y->elements, sizeof(float) * y->width * y->height, hipMemcpyHostToDevice));
	SAFE_CALL(hipMemcpy(gpu_params, Parameters->elements, sizeof(float) * Parameters->width * Parameters->height, hipMemcpyHostToDevice));

	// invoke kernel
	static const int blockWidth = 16;
	static const int blockHeight = blockWidth;
	int numBlocksW = X->width / blockWidth;
	int numBlocksH = X->height / blockHeight;
	if (X->width % blockWidth) numBlocksW++;
	if (X->height % blockHeight) numBlocksH++;

	dim3 dimGrid(numBlocksW, numBlocksH);
	dim3 dimBlock(blockWidth, blockHeight);

	dim3 dimReduce((m - 1) / REDUCE_BLOCK_SIZE + 1);
	dim3 dimReduceBlock(REDUCE_BLOCK_SIZE);

	dim3 dimVectorGrid(((m - 1) / blockWidth * blockWidth) + 1);
	dim3 dimVectorBlock(blockWidth * blockWidth);

	float* error_accum = new float[numOutputElements];
	for (int iter = 0; iter < maxIterations; ++iter) {
		for (int i = 0; i < m; ++i) {
			kernelMatrixMul<<<dimGrid, dimBlock>>>(&gpu_X[i * X->width], gpu_params, gpu_prediction, X->width, Parameters->width, 1, 1);
			kernelSigmoid<<<dimVectorGrid, dimVectorBlock>>>(gpu_prediction, 1);
			kernelUpdateParams<<<dimVectorGrid, dimVectorBlock>>>(gpu_prediction, &gpu_y[i], gpu_params, &gpu_X[i * X->width], Parameters->height, alpha);
		}
		kernelMatrixMul<<<dimGrid, dimBlock>>>(gpu_X, gpu_params, gpu_predictions, X->width, Parameters->width, predictions.width, predictions.height);
		kernelSigmoid<<<dimVectorGrid, dimVectorBlock>>>(gpu_predictions, m);


		// calculate error
		kernelAbsError<<<dimVectorGrid, dimVectorBlock>>>(gpu_predictions, gpu_y, gpu_abs_error, m);
		kernelReduction<<<dimReduce, dimReduceBlock>>>(gpu_abs_error, gpu_err_cost, m);
		SAFE_CALL(hipMemcpy(error_accum, gpu_err_cost, sizeof(float) * numOutputElements, hipMemcpyDeviceToHost));
		float g_sum = 0;
		for (int i = 0; i < numOutputElements; ++i)
		{
			g_sum += error_accum[i];
		}

		g_sum /= (2*m);

		cost_function.push_back(g_sum);
		sum += g_sum;
		quantity++;
		cout << g_sum << "\n";
	}

	mean_error = sum/quantity;
	printf("\n The mean error is %f\n", mean_error);

	cout << numOutputElements << "\n";

	int num_samples = y->height;
	cout << num_samples << "\n";
	calculate_confusion_matrix(gpu_predictions, gpu_y, d_confusion_matrix, num_classes, num_samples);

    // Allocate host memory for the confusion matrix and copy result back
    int h_confusion_matrix[4];
    SAFE_CALL(hipMemcpy(h_confusion_matrix, d_confusion_matrix, num_classes * num_classes * sizeof(int), hipMemcpyDeviceToHost));
	
	float tp = static_cast< float >(h_confusion_matrix[0]);
	float fp = static_cast< float >(h_confusion_matrix[1]);
	float fn = static_cast< float >(h_confusion_matrix[2]);
	float tn = static_cast< float >(h_confusion_matrix[3]);
	float acc = (tp+tn)/num_samples;
	float f1score = (2*tp)/(2*tp + fp + fn);

	std::cout << "Confusion Matrix:" << std::endl;
    for (int i = 0; i < num_classes; ++i) {
        for (int j = 0; j < num_classes; ++j) {
            std::cout << h_confusion_matrix[i * num_classes + j] << " ";
        }
        std::cout << std::endl;
    }

	std::cout << "Accuracy: " << acc << " ";
	std::cout << "F1 score: " << f1score;

	cout << endl;

	delete[] error_accum;
	SAFE_CALL(hipFree(gpu_X));
	SAFE_CALL(hipFree(gpu_y));
	SAFE_CALL(hipFree(gpu_abs_error));
	SAFE_CALL(hipFree(gpu_prediction));
	SAFE_CALL(hipFree(gpu_predictions));
	SAFE_CALL(hipFree(gpu_params));
	SAFE_CALL(hipFree(gpu_err_cost));
	SAFE_CALL(hipFree(d_confusion_matrix));
}

int main(int argc, char *argv[])
{
	string input_file = "";
	int num_classes=2;
	cout << "Please enter a valid file to run test for logistic regression on CUDA:\n>";
	getline(cin, input_file);
 	cout << "You entered: " << input_file << endl << endl;
    Matrix X,y;
    readData (input_file, &X, &y);
    cout <<"\n The X - Matrix." << endl;
    DisplayMatrix (X,true);
    cout <<"\n The y - Matrix." << endl;
    DisplayMatrix (y,true);

    Matrix Parameters, Train_Parameters;
    //Setup matrices with 1 as value initially
    AllocateMatrix(&Parameters, X.width, 1);
    AllocateMatrix(&Train_Parameters, X.width, 1);
    //Initialize with random +1 and -1 parameters.
    InitializeRandom(&Parameters, -1.0, 1.0);

    minMaxNormalization(&X);

    vector<float> cost_function;

    hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

    //unsigned int timer;
    //CUT_SAFE_CALL(cutCreateTimer(&timer));
    
    //cutStartTimer(timer);
    hipEventRecord(start);
    LogisticRegression(&X, &y, &Parameters, &Train_Parameters, 150, 0.03, cost_function, num_classes);
    //cutStopTimer(timer);
    hipEventRecord(stop);

	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	cout <<"\n The resulting parameters." << endl;
	DisplayMatrix (Parameters,true);

	//printf("\nProcessing time: %f (ms)\n", cutGetTimerValue(timer));
	printf("\nProcessing time: %f (ms)\n", milliseconds);

    std::cout << "Done" << std::endl;
	system("pause");

	return 0;
}