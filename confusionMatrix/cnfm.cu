#include <hip/hip_runtime.h>
#include <iostream>

__global__ void confusion_matrix_kernel(int* d_predictions, int* d_labels, int* d_confusion_matrix, int num_classes, int num_samples) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < num_samples) {
        int pred = d_predictions[idx];
        int label = d_labels[idx];

        if (pred < num_classes && label < num_classes) {
            atomicAdd(&d_confusion_matrix[label * num_classes + pred], 1);
        }
    }
}

void calculate_confusion_matrix(int* d_predictions, int* d_labels, int* d_confusion_matrix, int num_classes, int num_samples) {
    // Initialize confusion matrix to zero on the GPU
    hipMemset(d_confusion_matrix, 0, num_classes * num_classes * sizeof(int));

    // Launch kernel to compute confusion matrix
    int threadsPerBlock = 256;
    int blocksPerGrid = (num_samples + threadsPerBlock - 1) / threadsPerBlock;

    confusion_matrix_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_predictions, d_labels, d_confusion_matrix, num_classes, num_samples);

    // Synchronize to ensure all threads are done
    hipDeviceSynchronize();
}

int main() {
    const int num_samples = 28;
    const int num_classes = 2;

    // Allocate and initialize host memory
    int h_predictions[num_samples] = {1,0,1,0,1,0,1,0,1,1,1,1,1,1,1,1,1,0,0,0,0,0,0,1,1,1,0,0}; // Fill this with your data
    int h_labels[num_samples] = {1,0,1,0,1,0,1,1,0,1,1,1,1,1,1,1,1,0,1,0,0,0,0,0,0,1,1,1};      // Fill this with your data

    // Allocate device memory
    int* d_predictions;
    int* d_labels;
    int* d_confusion_matrix;

    hipMalloc(&d_predictions, num_samples * sizeof(int));
    hipMalloc(&d_labels, num_samples * sizeof(int));
    hipMalloc(&d_confusion_matrix, num_classes * num_classes * sizeof(int));

    // Copy data from host to device
    hipMemcpy(d_predictions, h_predictions, num_samples * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_labels, h_labels, num_samples * sizeof(int), hipMemcpyHostToDevice);

    // Calculate confusion matrix
    calculate_confusion_matrix(d_predictions, d_labels, d_confusion_matrix, num_classes, num_samples);

    // Allocate host memory for the confusion matrix and copy result back
    int h_confusion_matrix[num_classes * num_classes];
    hipMemcpy(h_confusion_matrix, d_confusion_matrix, num_classes * num_classes * sizeof(int), hipMemcpyDeviceToHost);

    // Print the confusion matrix
    std::cout << "Confusion Matrix:" << std::endl;
    for (int i = 0; i < num_classes; ++i) {
        for (int j = 0; j < num_classes; ++j) {
            std::cout << h_confusion_matrix[i * num_classes + j] << " ";
        }
        std::cout << std::endl;
    }

    // Free device memory
    hipFree(d_predictions);
    hipFree(d_labels);
    hipFree(d_confusion_matrix);

    return 0;
}
